#include "hip/hip_runtime.h"
#include "kernel_pca.h"
#include <stdio.h> /* for fprintf and stderr */





double* dev_fit_transform(hipblasHandle_t h, int M, int N, double *dR, int K)
{


	hipError_t status;

	// maximum number of iterations
	int J = 10000;

	// max error
	double er = 1.0e-7;

        // if no K specified, or K > min(M, N)
        int K_;
        K_ = min(M, N);
        if (K == -1 || K > K_) K = K_;

	

	int n, j, k;

	// allocate device memory for T, P
	double *dT = 0;
	status = hipMalloc(&dT, M*K*sizeof(dT[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dT)\n");
	}

	double *dP = 0;
	status = hipMalloc(&dP, N*K*sizeof(dP[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dP)\n");
	}

	// allocate memory for eigenvalues
	double *L;
	L = (double*)malloc(K * sizeof(L[0]));;
	if(L == 0)
	{
		fprintf(stderr, "! memory allocation error: T\n");
	}

	// mean center the data
	double *dU = 0;
	status = hipMalloc(&dU, M*sizeof(dU[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dU)\n");
	}

	hipblasDcopy(h, M, &dR[0], 1, dU, 1);

	double one = 1.0;
	double n_one = -1.0;
	for(n=1; n<N; n++)
	{
		hipblasDaxpy(h, M, &one, &dR[n*M], 1, dU, 1);
	}

	double neg_one_n = -1.0/N;

	for(n=0; n<N; n++)
	{
		hipblasDaxpy(h, M, &neg_one_n, dU, 1, &dR[n*M], 1);
	}

	double zero = 0.0;
	double *norm;
	double one_over_norm;	
	double one_Lk;
	double n_Lk;
	// GS-PCA
	double a;
	for(k=0; k<K; k++)
	{
		hipblasDcopy (h, M, &dR[k*M], 1, &dT[k*M], 1);
		a = 0.0;
		for(j=0; j<J; j++)
		{
			hipblasDgemv (h, HIPBLAS_OP_T, M, N, &one, dR, M, &dT[k*M], 1, &zero, &dP[k*N], 1);
			if(k>0)
			{
				hipblasDgemv (h, HIPBLAS_OP_T, N, k, &one, dP, N, &dP[k*N], 1, &zero, dU, 1);
				hipblasDgemv (h, HIPBLAS_OP_N, N, k, &n_one, dP, N, dU, 1, &one, &dP[k*N], 1);
			}
	
			hipblasDnrm2(h, N, &dP[k*N], 1, norm);	
			one_over_norm = 1.0/(*norm);
			hipblasDscal (h, N, &one_over_norm , &dP[k*N], 1);
			hipblasDgemv (h, HIPBLAS_OP_N, M, N, &one, dR, M, &dP[k*N], 1, &zero, &dT[k*M], 1);
			if(k>0)
			{
				hipblasDgemv (h, HIPBLAS_OP_T, M, k, &one, dT, M, &dT[k*M], 1, &zero, dU, 1);
				hipblasDgemv (h, HIPBLAS_OP_N, M, k, &n_one, dT, M, dU, 1, &one, &dT[k*M], 1);
			}

			hipblasDnrm2(h, M, &dT[k*M], 1, &L[k]);
			one_Lk = 1.0/L[k];
			hipblasDscal(h, M, &one_Lk, &dT[k*M], 1);

			if(fabs(a - L[k]) < er*L[k]) break;
			
			a = L[k];
			
		}
		n_Lk = - L[k];
			
		hipblasDger (h, M, N, &n_Lk, &dT[k*M], 1, &dP[k*N], 1, dR, M);

	}

	for(k=0; k<K; k++)
	{
		hipblasDscal(h, M, &L[k], &dT[k*M], 1);
	}

	// clean up memory
	free(L);
	status = hipFree(dP);
	status = hipFree(dU);

	return dT;

}



float* dev_fit_transform(hipblasHandle_t h, int M, int N, float *dR, int K)
{

	hipError_t status;

	// maximum number of iterations
	int J = 10000;

	// max error
	float er = 1.0e-7;

        // if no K specified, or K > min(M, N)
        int K_;
        K_ = min(M, N);
        if (K == -1 || K > K_) K = K_;

	int n, j, k;

	// allocate device memory for T, P
	float *dT = 0;
	status = hipMalloc(&dT, M*K*sizeof(dT[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dT)\n");
	}

	float *dP = 0;
	status = hipMalloc(&dP, N*K*sizeof(dP[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dP)\n");
	}

	// allocate memory for eigenvalues
	float *L;
	L = (float*)malloc(K * sizeof(L[0]));;
	if(L == 0)
	{
		fprintf(stderr, "! memory allocation error: T\n");
	}

	// mean center the data
	float *dU = 0;
	status = hipMalloc(&dU, M*sizeof(dU[0]));
	if(status != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "! cuda memory allocation error (dU)\n");
	}

	float one = 1.0;
	float n_one = -1.0;

	hipblasScopy(h, M, &dR[0], 1, dU, 1);
	for(n=1; n<N; n++)
	{
		hipblasSaxpy (h, M, &one, &dR[n*M], 1, dU, 1);
	}

	float neg_one_n = -1.0/N;
	for(n=0; n<N; n++)
	{
		hipblasSaxpy (h, M, &neg_one_n, dU, 1, &dR[n*M], 1);
	}
	
	float zero = 0.0;
	float *norm;
	float one_over_norm;	
	float one_Lk;
	float n_Lk;
	// GS-PCA
	float a;
	for(k=0; k<K; k++)
	{
		hipblasScopy (h, M, &dR[k*M], 1, &dT[k*M], 1);
		a = 0.0;
		for(j=0; j<J; j++)
		{
			hipblasSgemv (h, HIPBLAS_OP_T, M, N, &one, dR, M, &dT[k*M], 1, &zero, &dP[k*N], 1);
			if(k>0)
			{
				hipblasSgemv (h, HIPBLAS_OP_T, N, k, &one, dP, N, &dP[k*N], 1, &zero, dU, 1);
				hipblasSgemv (h, HIPBLAS_OP_N, N, k, &n_one, dP, N, dU, 1, &one, &dP[k*N], 1);
			}
			hipblasSnrm2(h, N, &dP[k*N], 1, norm);
			one_over_norm = 1.0/(*norm);
			hipblasSscal (h, N, &one_over_norm, &dP[k*N], 1);
			hipblasSgemv (h, HIPBLAS_OP_N, M, N, &one, dR, M, &dP[k*N], 1, &zero, &dT[k*M], 1);
			if(k>0)
			{
				hipblasSgemv (h, HIPBLAS_OP_T, M, k, &one, dT, M, &dT[k*M], 1, &zero, dU, 1);
				hipblasSgemv (h, HIPBLAS_OP_N, M, k, &n_one, dT, M, dU, 1, &one, &dT[k*M], 1);
			}

			hipblasSnrm2(h, M, &dT[k*M], 1, &L[k]);
			one_Lk = 1.0/L[k];
			hipblasSscal(h, M, &one_Lk, &dT[k*M], 1);

			if(fabs(a - L[k]) < er*L[k]) break;
			
			a = L[k];
			
		}
			
		n_Lk = - L[k];
		hipblasSger (h, M, N, &n_Lk, &dT[k*M], 1, &dP[k*N], 1, dR, M);
	

	}

	for(k=0; k<K; k++)
	{
		hipblasSscal(h, M, &L[k], &dT[k*M], 1);
	}

	// clean up memory
	free(L);
	status = hipFree(dP);
	status = hipFree(dU);

	return dT;

}






